#include "hip/hip_runtime.h"
/* 128 block with 128 thread for each block */
/* do N add calculations parallelly */

#include "hip/hip_runtime.h"
#include ""
#include <book.h>
#include <stdio.h>
#define N 10
/* grid�� ����� ������ ������ ���� (3����) */

__global__ void add(int * a, int * b, int * c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; //(������ �ε���) + (�� �ε���) * (�� ���� ������ ��)
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

	add << <128, 128 >> > (dev_a, dev_b, dev_c); //128 block, 128 thread
	
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	bool success = true;
	for (int i = 0; i < N; i++)
	{
		if (a[i] + b[i] != c[i])
		{
			printf("Error : %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
		if (success)
			printf("success\n");

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}
    return 0;
}
