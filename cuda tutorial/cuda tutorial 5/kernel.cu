#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <cpu_bitmap.h>
#include <book.h>
#include <time.h>

#define rnd(x) (x*rand()/RAND_MAX)
#define DIM 1024
#define SPHERES 40
#define INF 2e10f

struct Sphere
{
	/*
	���� �Ӽ���
	*/
	float r, g, b;
	float radius;
	float x, y, z;

	/*
	projective view
	ī�޶��� ��ġ�� +INF
	(ox, oy)�� ��ġ�� �ȼ����� �߻��� ������ ���� �浹�ϴ��� �Ǻ�. �浹�� �� ���̰� ��ȯ 
	*/
	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if (radius * radius > dx*dx + dy*dy)
		{
			float dz = sqrtf(radius*radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius*radius); //������ �ο��ϱ� ���� scale ��. ���� �߽ɿ��� �־������� �۾���
			return dz + z; //���̰��� ��ȯ
		}
		return -INF;
	}
};

__constant__ Sphere s[SPHERES];

__global__ void kernel(unsigned char *ptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x*gridDim.x;
	
	/*
	z���� ȭ���� ������ ������ ��
	*/
	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	/*
	�� ������(�ȼ�)���� ��� ������ �浹 �˻�
	�� ����� ���� �ȼ� ������ ����
	*/
	float r = 0, g = 0, b = 0;
	float maxz = -INF;
	for (int i = 0; i < SPHERES; i++)
	{
		float n = 0;
		float t = s[i].hit(ox, oy, &n);
		if (t > maxz)
		{
			float scale = n;
			r = s[i].r*scale;
			g = s[i].g*scale;
			b = s[i].b*scale;
			maxz = t;
		}
	}
	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

int main()
{
	/*
	���� ������ ���� event �Լ� ȣ��
	*/
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	srand(time(NULL));

	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	/*
	gpu �޸� �Ҵ�
	*/
	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
	//HANDLE_ERROR(hipMalloc((void**)&s, sizeof(Sphere)*SPHERES));

	/*
	�� �����͸� cpu �޸𸮿� ����
	*/
	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere)*SPHERES);
	for (int i = 0; i < SPHERES; i++)
	{
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(1000.0f) - 500;
		temp_s[i].y = rnd(1000.0f) - 500;
		temp_s[i].z = rnd(1000.0f) - 500;
		temp_s[i].radius = rnd(100.0f) + 20;
	}

	/*
	gpu �޸𸮷� �� ������ ����
	*/
	//HANDLE_ERROR(hipMemcpy(s, temp_s, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere)*SPHERES));
	free(temp_s);

	/*
	kernel ����
	*/
	dim3 grids(DIM / 16, DIM / 16);	//16*16��
	dim3 threads(16, 16);			//���� 16*16������
	kernel << <grids, threads >> > (dev_bitmap);

	/*
	gpu �޸𸮷κ��� ��Ʈ�� ����
	*/
	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time to generate : %3.3f ms\n", elapsedTime);
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	bitmap.display_and_exit();
    
	hipFree(dev_bitmap);
	return 0;
}