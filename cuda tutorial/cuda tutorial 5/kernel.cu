#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <cpu_bitmap.h>
#include <book.h>

#define rnd(x) (x*rand()/RAND_MAX)
#define DIM 1024
#define SPHERES 10
#define INF 2e10f

struct Sphere
{
	/*
	���� �Ӽ���
	*/
	float r, g, b;
	float radius;
	float x, y, z;

	/*
	projective view
	(ox, oy)�� ��ġ�� �ȼ����� �߻��� ������ ���� �浹�ϴ��� �Ǻ�. �浹�� �� ���̰� ��ȯ 
	*/
	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if (radius * radius > dx*dx + dy*dy)
		{
			float dz = sqrtf(radius*radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius*radius); //������ �ο��ϱ� ���� scale ��. ���� �߽ɿ��� �־������� �۾���
			return dz + z; //���̰��� ��ȯ
		}
		return -INF;
	}
};

Sphere *s;

int main()
{
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	HANDLE_ERROR(hipMalloc((void**)dev_bitmap, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&s, sizeof(Sphere)*SPHERES));

	/*
	�� �����͸� cpu �޸𸮿� ����
	*/
	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere)*SPHERES);
	for (int i = 0; i < SPHERES; i++)
	{
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(1000.0f) - 500;
		temp_s[i].y = rnd(1000.0f) - 500;
		temp_s[i].z = rnd(1000.0f) - 500;
		temp_s[i].radius = rnd(100.0f) + 20;
	}

    return 0;
}