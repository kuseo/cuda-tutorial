
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <book.h>
#include <cpu_bitmap.h>
#define DIM 512

/*
���� ������ �ڵ�
*/
GLuint bufferObj;
hipGraphicsResource *resource;

int main(int argc, char **argv)
{
	hipDeviceProp_t prop;	//cuda device
	int dev;	//cuda device �ĺ���
	
	/*
	cuda device ��� ���� 0���� �ʱ�ȭ�� �� �� ������ 1, �� ������ 0���� ����
	*/
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	

	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(cudaGLSetGLDevice(dev));

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("bitmap");

	//glutMainLoop();
    return 0;
}
